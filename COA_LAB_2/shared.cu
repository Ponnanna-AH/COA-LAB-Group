#include <iostream>
#include <cassert>
#include <ctime>
#include <cstdlib>
#include <hip/hip_runtime.h>

const int N = 128;
const int BLOCK_SIZE = 16;

__global__ void matrixMultiplyShared(float *A, float *B, float *C, int n)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;

    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.0f;

    for (int i = 0; i < n / BLOCK_SIZE; ++i)
    {
        shared_A[ty][tx] = A[row * n + i * BLOCK_SIZE + tx];
        shared_B[ty][tx] = B[(i * BLOCK_SIZE + ty) * n + col];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            sum += shared_A[ty][k] * shared_B[k][tx];
        }
        __syncthreads();
    }

    C[row * n + col] = sum;
}

void verifyMatrixMultiplication(float *A, float *B, float *C, int n)
{
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k)
            {
                sum += A[i * n + k] * B[k * n + j];
            }
            assert(fabs(C[i * n + j] - sum) < 1e-5);
        }
    }
}

int main()
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int matrixSize = N * N * sizeof(float);

    h_A = new float[N * N];
    h_B = new float[N * N];
    h_C = new float[N * N];

    srand(static_cast<unsigned int>(time(0)));
    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = static_cast<float>(rand() % 100 + 1);
        h_B[i] = static_cast<float>(rand() % 100 + 1);
    }

    hipMalloc(&d_A, matrixSize);
    hipMalloc(&d_B, matrixSize);
    hipMalloc(&d_C, matrixSize);

    hipMemcpy(d_A, h_A, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

    matrixMultiplyShared<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, matrixSize, hipMemcpyDeviceToHost);

    verifyMatrixMultiplication(h_A, h_B, h_C, N);

    std::cout << "Matrix multiplication result is correct." << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
